#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

#define BLOCK_DIM     16

typedef float dtype;

__global__
void naiveTrans(dtype *AT, dtype *A, int N) {
    
    int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int y = blockIdx.y * BLOCK_DIM + threadIdx.y;

    if(x < N && y < N)
        AT[x * N + y] = A[y * N + x];
    
}

__global__ 
void matTrans(dtype* AT, dtype* A, int N)  {

    __shared__ dtype block[BLOCK_DIM][BLOCK_DIM+1];

    int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int y = blockIdx.y * BLOCK_DIM + threadIdx.y;

    if(x < N && y < N)
        block[threadIdx.y][threadIdx.x] = A[y * N + x];

    __syncthreads();

    x = blockIdx.y * BLOCK_DIM + threadIdx.x;
    y = blockIdx.x * BLOCK_DIM + threadIdx.y;

    if(x < N && y < N)
	    AT[y * N + x] = block[threadIdx.x][threadIdx.y];
    
}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}


void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}

void
gpuTranspose (dtype* A, dtype* AT, int N)
{
    struct stopwatch_t* timer = NULL;
    long double t_gpu;

    hipError_t error;

    /* allocate data on GPU */
    dtype *A_gpu, *AT_gpu;
    error = hipMalloc((void **) &A_gpu, (N * N * sizeof (dtype)));
    if(error != hipSuccess)
    {
	    printf("hipMalloc failed: %d\n", error);
    }
    error = hipMalloc((void **) &AT_gpu, (N * N * sizeof (dtype)));
    if(error != hipSuccess)
    {
	    printf("hipMalloc failed: %d\n", error);
    }

    // copy host data to GPU
    error = hipMemcpy(A_gpu, A, N * N * sizeof (dtype), hipMemcpyHostToDevice);	
    if(error != hipSuccess)
    {
        printf("hipMemcpy failed: %d\n", error);
    } 
    error = hipMemset(AT_gpu, 0, N * N * sizeof(dtype));
    if(error != hipSuccess)
    {
        printf("hipMemset failed: %d\n", error);
    }

    dim3 grid(N/BLOCK_DIM, N/BLOCK_DIM, 1), threads(BLOCK_DIM, BLOCK_DIM, 1);

    /* Setup timers */
    stopwatch_init ();
    timer = stopwatch_create ();

    

    /* warmup */
    matTrans<<<grid, threads>>>(AT_gpu, A_gpu, N);
    error = hipDeviceSynchronize();
    printf("hipDeviceSynchronize error code: %d\n", error);
    
    int numIterations = 1;
    /* run transpose kernel */
    for(int i = 0; i < numIterations; i++)
    {
        stopwatch_start (timer);
	
        matTrans<<<grid, threads>>>(AT_gpu, A_gpu, N);

        error = hipDeviceSynchronize ();
        printf("hipDeviceSynchronize error code: %d\n", error);
        t_gpu = stopwatch_stop (timer);
        fprintf (stderr, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",               
            t_gpu, (N * N) / t_gpu * 1e-9 );
    }

    /* copy results from GPU back to host */
    error = hipMemcpy(AT, AT_gpu, N * N * sizeof (dtype), hipMemcpyDeviceToHost);
    if(error != hipSuccess)
    {
	    printf("hipMemcpy (copying back from device to host) failed: %d\n", error);
    }

    hipFree(A_gpu);
    hipFree(AT_gpu);
}

int 
main(int argc, char** argv)
{
    /* variables */
    dtype *A, *ATgpu, *ATcpu;
    int err;

	int N;

    struct stopwatch_t* timer = NULL;
    long double t_cpu;


	N = -1;
	parseArg (argc, argv, &N);

    /* input and output matrices on host */
    /* output */
    ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
    ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

    /* input */
    A = (dtype*) malloc (N * N * sizeof (dtype));

    initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

    /* Setup timers */
    stopwatch_init ();
    timer = stopwatch_create ();

	stopwatch_start (timer);
    
    /* compute reference array */	
    cpuTranspose (A, ATcpu, N);
    t_cpu = stopwatch_stop (timer);
    fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

    /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
